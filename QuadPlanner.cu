#include "hip/hip_runtime.h"
#include <math.h>
#include "Structs.h"
#include <vector>
#include <stdlib.h>
#include <stdio.h>
#include "hash_map_template.h"
#include "hip/hip_runtime_api.h"
 
#define	STARTING_VALUE -1
#define OBSTACLE_VALUE -2
#define GOAL_VALUE -3
#define INVALID_QUAD -5.0f
#define NUM_NEIGH_PER_QUAD	20	
#define BLOCK_SIZE 512
//Defines Texture and its methods

using namespace CUDASTL;

HashMap<int, int>* hashmap;
NeighborStruct* neighborsDev;

__device__ void neighborsForQuadDev(NeighborStruct* neighbors, QuadStruct* quad, HashMap<int, int> *hashmap);

__device__ bool equals(float x1, float x2)
{
	if (fabs(x1 - x2) < .01) {
		return true;
	}
	return false;
}

__device__ float distance(QuadStruct *from, QuadStruct *to) {
	
	if (equals(from->centerx,to->centerx)) {
		return fabs(to->centery - from->centery);
	}
	else if(equals(from->centery,to->centery)) {
		return fabs(to->centerx - from->centerx);
	}
	else {
		return sqrt(fabs(to->centerx - from->centerx)*fabs(to->centerx - from->centerx) +
			fabs(to->centery - from->centery)*fabs(to->centery - from->centery));
	}
}

__device__ int stateNeedsUpdate(QuadStruct* state) {
	return state->g == STARTING_VALUE || state->g == GOAL_VALUE;
}

__device__ int stateIsObstacle(QuadStruct* state) {
	return state->costToReach > 10.0f;
}

__device__ int QisGoalState(QuadStruct* state) {
	return state->g == 0.0f;
}

//Kernel function for planner

__global__ void computeNeighborsKernel(QuadStruct *current_texture, HashMap<int, int> *hashmap, int numberOfQuads, NeighborStruct* neighbors)
{
	int id = get_thread_id();
	if (id < numberOfQuads) {
		QuadStruct quad = current_texture[id];
		if (quad.g != INVALID_QUAD) {
			neighborsForQuadDev(&neighbors[id*NUM_NEIGH_PER_QUAD], &quad, hashmap);
			current_texture[id] = quad;
		}
	}
}

extern "C" void computeNeighbors(QuadStruct* texture, int numberOfQuads)
{
	int gridLength = ceil((double)numberOfQuads/(double)BLOCK_SIZE);
	
	dim3 blocks(gridLength, 1, 1);
	dim3 threads(BLOCK_SIZE, 1, 1);
	
	QuadStruct *texture_device;
	hipMalloc((void**)&texture_device, (numberOfQuads)*sizeof(QuadStruct));
	
	//make a two copies of the initial map
	hipMemcpy(texture_device, texture, (numberOfQuads)*sizeof(QuadStruct), hipMemcpyHostToDevice);

	hipFree(neighborsDev);
	hipMalloc((void**)&neighborsDev, (numberOfQuads*NUM_NEIGH_PER_QUAD)*sizeof(NeighborStruct));
	hipMemset(neighborsDev, -1, (numberOfQuads*NUM_NEIGH_PER_QUAD)*sizeof(NeighborStruct));

	computeNeighborsKernel<<<blocks, threads>>>(texture_device, hashmap, numberOfQuads, neighborsDev);

	hipMemcpy(texture, texture_device, (numberOfQuads)*sizeof(QuadStruct), hipMemcpyDeviceToHost);
	hipFree(texture_device);
}

__global__ void QcomputeCostsKernel(QuadStruct *current_texture, QuadStruct *texture_copy, NeighborStruct* neighbors, int numberOfQuads, int *check, int *locality, float maxCost, bool allAgentsReached) {
	int id = get_thread_id();

	if (id < numberOfQuads) {
		QuadStruct quad = current_texture[id];

		//if(!stateIsObstacle(state) && !isGoalState(state)) {
			//if the state is an obstacle, do not compute neighbors
		if (!QisGoalState(&quad) && quad.g != INVALID_QUAD) {

			int predecesorIndex;
			float originalG = quad.g;
			quad.g = STARTING_VALUE;
			for (int i = 0; i < NUM_NEIGH_PER_QUAD; ++i) {
				NeighborStruct neighbor_struct = neighbors[(id*NUM_NEIGH_PER_QUAD)+i];
				if (neighbor_struct.indexInMap < 0 ) 
					break;
				QuadStruct neighbor = texture_copy[neighbor_struct.indexInMap]; //Needs to find a quad in the ro map
				
				//if (neighbor.quadCode != neighbor_struct.quadCode) {
					//...Our index in map is outdated and neighbors need to be recomputed for this quad. Determine best way to go about it.
				//}

				if (stateIsObstacle(&neighbor)) //if neighbor is an obstacle, do not use it as a possible neighbor
					continue;
				float newg = neighbor.g + distance(&neighbor, &quad) * quad.costToReach;
				if ((newg < quad.g || stateNeedsUpdate(&quad)) && !stateNeedsUpdate(&neighbor)) {
					predecesorIndex = neighbor.indexInMap;
					quad.prevQuadCode = neighbor.quadCode;
					quad.g = newg;
				}

			
		/*	QuadStruct *selectedPredecessorCopy = &texture_copy[predecesorIndex];
			quad->inconsistent = false;
			//if ((selectedPredecessorCopy != NULL && selectedPredecessorCopy->inconsistent) || stateIsObstacle(selectedPredecessorCopy)) {
			if (selectedPredecessorCopy->inconsistent) {
				//if predecessor from read-only is inconsistent - clear inconsistent flag in write-only and mark state as inconsistent in write-only
				current_texture[predecesorIndex].inconsistent = false;
				quad->inconsistent = true;
				quad->g = STARTING_VALUE;
			} */
			}
			if (*locality == 1 && originalG != quad.g) {
				*check = 0;
			} else if (*locality == 2) {
				if ((originalG != quad.g && quad.g < maxCost) || !allAgentsReached) {
					*check = 0;
				}
			} else if (*locality == 0 && allAgentsReached && originalG != quad.g) {
				*check = 1;
			}
		}

		current_texture[id] = quad;
	}	
}

__global__ void checkForInconsistency(QuadStruct* texture, int numberOfQuads, int* flag) {
	int id = get_thread_id();

	if (id < numberOfQuads) {
		QuadStruct* state = &texture[id];
		if (state->inconsistent) {
			*flag = 1;
		}
	}
}

float agentsMaxCost(QuadStruct* texture, int agentCount, QuadStruct* agents, int goalNumber) {
	float maxCost = -10000.0f;
	for (int i = 0; i < agentCount; i++)  {
		QuadStruct agent = texture[agents[i].indexInMap];
		if (agent.g > maxCost) {
			maxCost = agent.g;
		}
	}
	return maxCost;
}

bool agentsReached(QuadStruct* texture, int agentCount, QuadStruct* agents, int goalNumber) {
	for (int i = 0; i < agentCount; i++) {
		QuadStruct agent = texture[agents[i].indexInMap];
		if (agent.g < 0.0f) {
			return false;
		}
	}
	return true;
}


extern "C" int QcomputeCostsCuda(QuadStruct* texture, int numberOfQuads, int locality, int agentCount, QuadStruct* agents, int goalNumber) {
	int *locality_dev, *consistencyCheck, *consistencyCheck_dev, *flag, *flag_dev;
	
	int gridLength = ceil((double)numberOfQuads/(double)BLOCK_SIZE);
	
	dim3 blocks(gridLength, 1, 1);
	dim3 threads(BLOCK_SIZE, 1, 1);

	QuadStruct *texture_device, *texture_device_copy;
	hipMalloc((void**)&texture_device, (numberOfQuads)*sizeof(QuadStruct));
	hipMalloc((void**)&texture_device_copy, (numberOfQuads)*sizeof(QuadStruct));
	//make a two copies of the initial map
	hipMemcpy(texture_device, texture, (numberOfQuads)*sizeof(QuadStruct), hipMemcpyHostToDevice);
	hipMemcpy(texture_device_copy, texture, (numberOfQuads)*sizeof(QuadStruct), hipMemcpyHostToDevice);

	hipMalloc((void**)&locality_dev, sizeof(int));
	int* locality_ptr = (int*)malloc(sizeof(int));
	*locality_ptr = locality;

	hipMalloc((void**)&consistencyCheck_dev, sizeof(int));
	consistencyCheck = (int*)malloc(sizeof(int));

	hipMalloc((void**)&flag_dev, sizeof(int));
	flag = (int*)malloc(sizeof(int));

	int iterations = 0;

	do {
		//set flag to 0 to check for changes
		if (locality == 1 || locality == 2) {
			*consistencyCheck = 1;
		} else {
			*consistencyCheck = 0;
		}
		*flag = 0;
		hipMemcpy(locality_dev, locality_ptr, sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(consistencyCheck_dev, consistencyCheck, sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(flag_dev, flag, sizeof(int), hipMemcpyHostToDevice);

		bool allAgentsReached = agentsReached(texture, agentCount, agents, goalNumber);
		float maxCost;
		if (allAgentsReached) {
			maxCost = agentsMaxCost(texture, agentCount, agents, goalNumber);
		}

		QcomputeCostsKernel<<<blocks, threads>>>(texture_device, texture_device_copy, neighborsDev, numberOfQuads, consistencyCheck_dev, locality_dev, maxCost, allAgentsReached);
		
		checkForInconsistency<<<blocks, threads>>>(texture_device, numberOfQuads, flag_dev);
		
		QuadStruct* temp = texture_device;
		texture_device = texture_device_copy;
		texture_device_copy = temp;
		iterations++;
		
		hipMemcpy(texture, texture_device, (numberOfQuads)*sizeof(QuadStruct), hipMemcpyDeviceToHost);
		
		hipMemcpy(consistencyCheck, consistencyCheck_dev, sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(flag, flag_dev, sizeof(int), hipMemcpyDeviceToHost);
	} while(*consistencyCheck == 0 || *flag == 1);

	
	hipFree(texture_device); hipFree(texture_device_copy);

	printf("Result was: %i\n\n", *consistencyCheck);
	printf("Number of iterations: %i\n\n", iterations);

	return 1;
}

__global__ void propagateUpdateKernel(QuadStruct* texture, QuadStruct* texture_copy, int numberOfQuads, int* propagateUpdate, HashMap<int, int>* hashmap)
{
	int id = get_thread_id();
	if (id < numberOfQuads) {
		QuadStruct quad = texture_copy[id];
		if (quad.prevQuadCode > 0) {
			int predecessorIndex = *((*hashmap).valueForKey(quad.prevQuadCode));
			QuadStruct predecessor = texture[predecessorIndex];
				if (predecessor.g == STARTING_VALUE) {
					texture[id].g = STARTING_VALUE;
					texture[id].prevQuadCode = 0;
					*propagateUpdate = 1;
				}
			}
	}
}

extern "C" void propagateUpdateAfterObstacleMovement(QuadStruct* texture, int numberOfQuads)
{
	int gridLength = ceil((double)numberOfQuads/(double)BLOCK_SIZE);

	dim3 blocks(gridLength, 1, 1);
	dim3 threads(BLOCK_SIZE, 1, 1);

	QuadStruct* texture_dev, *texture_copy_dev;
	hipMalloc((void**)&texture_dev, sizeof(QuadStruct)*numberOfQuads);
	hipMalloc((void**)&texture_copy_dev, sizeof(QuadStruct)*numberOfQuads);

	hipMemcpy(texture_dev, texture, sizeof(QuadStruct)*numberOfQuads, hipMemcpyHostToDevice);
	hipMemcpy(texture_copy_dev, texture, sizeof(QuadStruct)*numberOfQuads, hipMemcpyHostToDevice);

	int* propagateUpdate = (int*)malloc(sizeof(int));
	
	int* propagateUpdate_dev;
	hipMalloc((void**)&propagateUpdate_dev, sizeof(int));
	do  {
		*propagateUpdate = 0;
		hipMemcpy(propagateUpdate_dev, propagateUpdate, sizeof(int), hipMemcpyHostToDevice);
		
		propagateUpdateKernel<<<blocks, threads>>>(texture_dev, texture_copy_dev, numberOfQuads,propagateUpdate_dev, hashmap);
		hipMemcpy(propagateUpdate, propagateUpdate_dev, sizeof(int), hipMemcpyDeviceToHost);
		QuadStruct* temp = texture_dev;
		texture_dev = texture_copy_dev;
		texture_copy_dev = texture_dev;

	} while (*propagateUpdate == 1);
	
	hipMemcpy(texture, texture_dev, sizeof(QuadStruct)*numberOfQuads, hipMemcpyDeviceToHost);
	hipFree(texture_dev); hipFree(texture_copy_dev);
	
}

__global__ void clearTextureValuesKernel(QuadStruct* texture, int numberOfQuads, int goalNumber) {
	int id = get_thread_id();

	if (id < numberOfQuads) {
		QuadStruct* state = &texture[id];
		state->g = STARTING_VALUE;
		state->prevQuadCode = 0;
		state->inconsistent = false;
	}

}

extern "C" void clearTextureValuesQuad(QuadStruct* texture, int numberOfQuads, int goalNumber) {
	QuadStruct* texture_dev;

	int gridLength = ceil((double)numberOfQuads/(double)BLOCK_SIZE);
	dim3 blocks(gridLength, 1, 1);
	dim3 threads(BLOCK_SIZE, 1, 1);

	hipMalloc((void**)&texture_dev, (numberOfQuads*sizeof(QuadStruct)));
	hipMemcpy(texture_dev, texture, numberOfQuads*sizeof(QuadStruct), hipMemcpyHostToDevice);
	clearTextureValuesKernel<<<blocks, threads>>> (texture_dev, numberOfQuads, goalNumber);
	hipMemcpy(texture, texture_dev, (numberOfQuads)*sizeof(QuadStruct), hipMemcpyDeviceToHost);

	hipFree(texture_dev);
}



/************************************************
******** HashMap Methods ************************
************************************************/

__global__ void populateHashMap(HashMap<int, int> *hash,  QuadStruct *quads, int numberOfQuads)
{
	int i = get_thread_id();
	if (i < numberOfQuads) {
		(*hash)[quads[i].quadCode] = quads[i].indexInMap;
	}
}

extern "C" void createHashMap(QuadStruct quads[], int numberOfQuads, int size)
{
	hashmap = CreateHashMap<int,int, HashFunc<int> >(ceil((double)numberOfQuads/32), size);
	hipError_t err = hipGetLastError();
	int blocks = ceil((double)numberOfQuads/512);

	QuadStruct* quads_dev, *q;
	q = (QuadStruct*)malloc(sizeof(QuadStruct));
	q = quads;

	hipMalloc((void **)&quads_dev, sizeof(QuadStruct)*numberOfQuads);
	hipMemcpy(quads_dev, q, sizeof(QuadStruct)*numberOfQuads, hipMemcpyHostToDevice);

	populateHashMap<<<blocks, 512>>> (hashmap, quads_dev, numberOfQuads);
	hipFree(quads_dev);
}

__global__ void retrieveQuadStruct(int* quad, int code, HashMap<int, int> *hashmap)
{
	int i = get_thread_id();
	if (i == 1) {
		int q = *((*hashmap).valueForKey(code));
		*quad = q; 
	}
}

extern "C" int *quadForCode(int code)
{
	int *q_dev, *q;
	q = (int*)malloc(sizeof(int));
	hipMalloc((void**)&q_dev, sizeof(int));
	
	retrieveQuadStruct<<<1, 512>>> (q_dev, code, hashmap);
	hipMemcpy(q, q_dev, sizeof(int), hipMemcpyDeviceToHost);
	hipFree(q_dev);
	return q;
}

extern "C" void cleanupDevice()
{
	DestroyHashMap(hashmap);
	hipDeviceReset();
}

__global__ void invalidateQuadsKernel(QuadStruct* quads, int* indexes, HashMap<int, int>* hashmap)
{
	int id = get_thread_id();
	//As many threads as objects to insert
	int* indexInMap = (*hashmap).valueForKey(quads[id].quadCode);
	int index = *indexInMap;
	indexes[id] = index;
	*indexInMap = -1;
}

extern "C" int* invalidateQuadsInHash(QuadStruct quadsRemoved[], int countRemoved)
{
	QuadStruct* ptr = quadsRemoved;
	QuadStruct* quads_dev;
	int* indexes = (int*)malloc(sizeof(int)*countRemoved);
	int* indexes_dev;

	hipMalloc((void**)&quads_dev, sizeof(QuadStruct)*countRemoved);
	hipMalloc((void**)&indexes_dev, sizeof(int)*countRemoved);

	hipMemcpy(quads_dev, ptr, sizeof(QuadStruct)*countRemoved, hipMemcpyHostToDevice);

	invalidateQuadsKernel<<<1, countRemoved>>>(quads_dev, indexes_dev, hashmap);

	hipMemcpy(indexes, indexes_dev, sizeof(int)*countRemoved, hipMemcpyDeviceToHost);

	hipFree(indexes_dev); hipFree(quads_dev);

	return indexes;
}

__global__ void updateQuadsKernel(QuadStruct* quads, int* indexes, HashMap<int, int>* hashmap)
{
	int id = get_thread_id();
	//As many threads as objects to insert
	int* indexInMap = (*hashmap).valueForKey(quads[id].quadCode);
	indexes[id] = *indexInMap;
}

extern "C" int* updateQuadsInHash(QuadStruct updateQuads[], int count)
{
	QuadStruct* ptr = updateQuads;
	QuadStruct* quads_dev;
	int* indexes = (int*)malloc(sizeof(int)*count);
	int* indexes_dev;

	hipMalloc((void**)&quads_dev, sizeof(QuadStruct)*count);
	hipMalloc((void**)&indexes_dev, sizeof(int)*count);

	hipMemcpy(quads_dev, ptr, sizeof(QuadStruct)*count, hipMemcpyHostToDevice);

	updateQuadsKernel<<<1, count>>>(quads_dev, indexes_dev, hashmap);

	hipMemcpy(indexes, indexes_dev, sizeof(int)*count, hipMemcpyDeviceToHost);

	hipFree(indexes_dev); hipFree(quads_dev);

	return indexes;
}

__global__ void insertNewQuadsKernel(QuadStruct quads[], HashMap<int, int>* hashmap)
{
	int id = get_thread_id();
	QuadStruct quad = quads[id];
	//As many threads as objects to insert
	int* indexInMap = (*hashmap).valueForKey(quad.quadCode);
	if (indexInMap == NULL) {
		(*hashmap)[quad.quadCode] = quad.indexInMap;
	} else {
		*indexInMap = quad.indexInMap;
	}
}

extern "C" void insertNewQuadsInHash(QuadStruct quadsInserted[], int countInserted)
{
	QuadStruct* ptr = quadsInserted;
	QuadStruct* quads_dev;

	hipMalloc((void**)&quads_dev, sizeof(QuadStruct)*countInserted); 
	hipMemcpy(quads_dev, ptr, sizeof(QuadStruct)*countInserted, hipMemcpyHostToDevice);
	
	insertNewQuadsKernel<<<1, countInserted>>>(quads_dev, hashmap);

	hipFree(quads_dev);
}


extern "C" void updateNeighborsToQuads(int* indexes, int size, int totalNumberOfQuads, QuadStruct** texture, int numberOfGoals, int* updateIndexes, int updateCount)
{
	NeighborStruct* neighbors = (NeighborStruct*) malloc ((totalNumberOfQuads*NUM_NEIGH_PER_QUAD)*sizeof(NeighborStruct));
	hipMemcpy(neighbors, neighborsDev, (totalNumberOfQuads*NUM_NEIGH_PER_QUAD)*sizeof(NeighborStruct), hipMemcpyDeviceToHost);

	int totalCount = size+updateCount; 
	for (int i = 0; i < totalCount; i++) {
		int offset = (i < size) ? indexes[i]*NUM_NEIGH_PER_QUAD : updateIndexes[i-size]*NUM_NEIGH_PER_QUAD;
		for (int j = 0; j < NUM_NEIGH_PER_QUAD; j++) {
			int neighborIndex = neighbors[offset+j].indexInMap;
			if (neighborIndex < 0) {
				continue;
			} else {
				for (int m = 0; m < numberOfGoals; m++) {
					texture[m][neighborIndex].g = STARTING_VALUE;
				}
			}
		}
	}
	free(neighbors);
}



/**************************************************************
******************** Neighbor Finding *************************
***************************************************************/

__device__ int constructNeighborQuadCode(QuadStruct* quad, int max_i, int codeDiff)
{		
	int code = 0;
	int codeDigit = quad->quadCode;
	int compareInt = 10, substract;

	int length = 1;
	while (codeDigit >= compareInt) {
		codeDigit /= 10;
		length++;
	}

	substract = codeDigit;

	for (int i = 0; i < length; i++) 
	{
		if (i+1 > max_i) {
			codeDigit -= codeDiff;	
		} else if (i+1 == max_i) {
			codeDigit += codeDiff;
		}

		code = (code*10) + codeDigit;

		compareInt *= 10;
		codeDigit = quad->quadCode;

		while (codeDigit >= compareInt) {
			codeDigit /= 10;
		}

		codeDigit -= substract*10;
		substract = substract*10+codeDigit;
	}
	return code;
}

__device__ int* greaterNeighbor(QuadStruct quad, int neighborQuadCode, HashMap<int, int> *hashmap, int* neighborCodes)
{
	int parsedCode = neighborQuadCode;

	int* neighbor = NULL;
	while (parsedCode > 0)
	{
		if (parsedCode == neighborQuadCode) {
			neighbor = (*hashmap).valueForKey(parsedCode);
			if (neighbor != NULL) {
				neighborCodes[0] = parsedCode;
				break;
			}
		}

		parsedCode = parsedCode/10;
		neighborQuadCode = neighborQuadCode/10;
	}

	return neighbor;
}

__device__ void populateSmallerNeighbors(QuadStruct* quad, int* neighbors, int quadCode, int addCode1, int addCode2, HashMap<int, int> *hashmap, int *indexOffset, int* neighborCodes)
{
	int* neighbor = (*hashmap).valueForKey(quadCode);

	if (neighbor != NULL && *neighbor != -1) {
		int q = *neighbor;
		neighbors[*indexOffset] = q;
		neighborCodes[*indexOffset] = quadCode;
		*indexOffset += 1;
		quad->neighborCount += 1;
	} else {
		int *codes = (int*)malloc(sizeof(int)*(16));
		codes[0] = (quadCode*10) + addCode1;
		codes[1] = (quadCode*10) + addCode2;
		int lastIndex = 2;
		for (int i = 0; i < lastIndex; i++) {
			int code = codes[i];
			if (code/1000000 > 5) { continue; }

			int* subNeighbor = (*hashmap).valueForKey(code);
			if (subNeighbor != NULL && *subNeighbor != -1) {
				int q = *subNeighbor;
				neighbors[*indexOffset] = q;
				neighborCodes[*indexOffset] = code;
				*indexOffset += 1;
				quad->neighborCount += 1;
			} else {
				codes[lastIndex] = (code*10) + addCode1;
				codes[lastIndex+1] = (code*10) + addCode2;
				lastIndex += 2;
			}
		}
	}
}

__device__ int* smallerNeighbors(QuadStruct* quad, int neighborQuadCode, int addCode1, int addCode2, HashMap<int, int> *hashmap, int* neighborCodes)
{
	int* neighbors = (int*)malloc(sizeof(int)*16);

	int neighborCode1 = (neighborQuadCode*10) + addCode1;
	int neighborCode2 = (neighborQuadCode*10) + addCode2;

	int currentNeighbors = quad->neighborCount;
	int *indexOffset = (int*)malloc(sizeof(int));
	*indexOffset = 0;
	populateSmallerNeighbors(quad, neighbors, neighborCode1, addCode1, addCode2, hashmap, indexOffset, neighborCodes);

	populateSmallerNeighbors(quad, neighbors, neighborCode2, addCode1, addCode2, hashmap, indexOffset, neighborCodes);

	return neighbors;
}

__device__ void retrieveNeighbors(NeighborStruct* neighbors, QuadStruct* quad, int max_i, HashMap<int, int> *hashmap, int codeDiff, int addCode1, int addCode2)
{
	int neighborCode = constructNeighborQuadCode(quad, max_i, codeDiff);
	int* neighborQuads = (*hashmap).valueForKey(neighborCode);
	int* neighborCodes = (int*) malloc(sizeof(int)*8);
	int startingNeighborCount = quad->neighborCount;
	bool freeMem = false;
	if (neighborQuads == NULL || *neighborQuads == -1) {
		neighborQuads = greaterNeighbor(*quad, neighborCode, hashmap, neighborCodes);
		if (neighborQuads == NULL || *neighborQuads == -1) {
			neighborQuads = smallerNeighbors(quad, neighborCode, addCode1, addCode2, hashmap, neighborCodes);
			freeMem = true;
		} else {
			quad->neighborCount += 1;
		}
	} else {
		quad->neighborCount += 1;
		neighborCodes[0] = neighborCode;
	}

	int diff = quad->neighborCount - startingNeighborCount;
	int indexForCode = 0;
	for (int i = 0; i < quad->neighborCount; i++)
	{
		if (neighbors[i].indexInMap < 0) {
			NeighborStruct neighbor;
			neighbor.indexInMap = neighborQuads[diff-1];
			neighbor.quadCode = neighborCodes[indexForCode];
			neighbors[i] = neighbor;
			diff--;
			indexForCode++;
			if (diff == 0) { break;}
		}
	}
	if (freeMem) {
		free(neighborQuads);
	}
	free(neighborCodes);
}

__device__ void neighborsForQuadDev(NeighborStruct* neighbors, QuadStruct* quad, HashMap<int, int> *hashmap)
{
	short int Emax_i = 0;  short int Wmax_i = 0; short int Nmax_i = 0; short int Smax_i = 0;
	short int length = 1;
	int codeDigit = quad->quadCode;
	int compareInt = 10;
	int substract = 0;
	quad->neighborCount = 0;
	
	while (codeDigit >= compareInt) {
			codeDigit /= 10;
			length++;
	}


	for (int i = 0; i < length; i++) 
	{
		while (codeDigit >= compareInt) {
			codeDigit /= 10;
		}
		codeDigit -= substract*10;
		substract = substract*10+codeDigit;

		if (codeDigit == 1 || codeDigit == 3) {
			Emax_i = i+1;	
		}

		if (codeDigit == 2 || codeDigit == 4) {
			Wmax_i = i+1;	
		}

		if (codeDigit == 1 || codeDigit == 2) {
			Smax_i = i+1;	
		}

		if (codeDigit == 3 || codeDigit == 4) {
			Nmax_i = i+1;	
		}

		compareInt *= 10;
		codeDigit = quad->quadCode;
	}

	if (Emax_i > 0) {
		retrieveNeighbors(neighbors, quad, Emax_i, hashmap, 1, 1, 3);
	}


	if (Wmax_i > 0) {
		retrieveNeighbors(neighbors, quad, Wmax_i, hashmap, -1, 2, 4);
	}


	if (Smax_i > 0) {
		retrieveNeighbors(neighbors, quad, Smax_i, hashmap, 2, 1, 2);
	}

	if (Nmax_i > 0) {
		retrieveNeighbors(neighbors, quad, Nmax_i, hashmap, -2, 3, 4);
	}
}